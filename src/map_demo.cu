#include <iostream>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <ctime>

using namespace std;

__global__  void addTen(float *d, int count)
{
    int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;

    int threadPosInBlock = threadIdx.x
                           + blockDim.x * threadIdx.y
                           + blockDim.x * blockDim.y * threadIdx.z;

    int blockPosInGrid = blockIdx.x
                         + gridDim.x * blockIdx.y
                         + gridDim.x * gridDim.y * blockIdx.z;

    int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;
    if (tid < count)
    {
        d[tid] = d[tid] + 10;
    }
}

int main()
{
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetPseudoRandomGeneratorSeed(gen, time(0));

    int const count = 123456;
    int const size = count * sizeof(float);
    float *d;
    float h[count];
    hipMalloc(&d, size);
    hiprandGenerateUniform(gen, d, count);

    dim3 block(8, 8, 8);
    dim3 grid(16, 16);

    addTen<<<grid, block>>>(d, count);
    hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    for(int i =0; i< 100; ++i)
    {
        cout<< h[i] <<" ";
    }
    cout<<endl;
    hipFree(d);


    return 0;
}