
#include <iostream>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <ctime>

using namespace std;

__global__ void sumSingleBlock(int *d)
{
    auto tid = threadIdx.x;

    for (int tc = blockDim.x, stepSize = 1; tc > 0; tc >>= 1, stepSize <<= 1)
    {
        if (tid < tc)
        {
            int pa = tid * stepSize * 2;
            int pb = pa + stepSize;
            d[pa] += d[pb];
        }
    }
}

int main()
{
    int const count = 512;
    int const size = count * sizeof(int);
    int h[count];

    for (int i = 0; i < count; ++i)
    {
        h[i] = i + 1;
    }

    int *d;
    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    sumSingleBlock<<<1, count / 2>>>(d);

    int res;
    hipMemcpy(&res, d, sizeof(int), hipMemcpyDeviceToHost);
    cout<<"Sum is: "<<res<<endl;
    hipFree(d);
    return 0;
}