#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

int main()
{
    int cont;
    hipGetDeviceCount(&cont);
    cout << cont << endl;
    hipDeviceProp_t prop;
    for (int i = 0; i < cont; ++i)
    {
        hipGetDeviceProperties(&prop, i);
        cout << "Device name: " << prop.name << endl;
        cout << "Compute capabilities: " << prop.major << "." << prop.minor << endl;

        cout << "Max grid dimensions (" << prop.maxGridSize[0] << " X " << prop.maxGridSize[1] << " X "
             << prop.maxGridSize[2] << ")\n";

        cout << "Max block dimensions (" << prop.maxThreadsDim[0] << " X " << prop.maxThreadsDim[1] << " X "
             << prop.maxThreadsDim[2] << ")\n";

    }
    return 0;
}