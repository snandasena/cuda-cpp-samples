#include <iostream>
#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <ctime>

using namespace std;

__global__ void runningSum(int *d)
{
    int threads = blockDim.x;
    int tid = threadIdx.x;

    for (int tc = threads, step = 1; tc > 0; step *= 2)
    {
        if (tid < tc)
        {
            d[tid + step] += d[tid];
        }
        tc -= step;
    }

}

int main()
{
    int const count = 16;
    int const size = count * sizeof(int);
    int h[count];
    for (int i = 0; i < count; ++i)
    {
        h[i] = i + 1;
    }

    int *d;
    hipMalloc(&d, size);
    hipMemcpy(d, h, size, hipMemcpyHostToDevice);

    runningSum<<<1, count - 1>>>(d);
    hipMemcpy(h, d, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < count; ++i)
    {
        cout << h[i] << "\t";
    }

    hipFree(d);

    return 0;
}