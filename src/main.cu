#include <iostream>
#include <hip/hip_runtime.h>


__global__ void addArr(int const *a, int const *b, int *c) // CUDA kernel
{
    auto i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
    int const cont = 5;
    int const size = cont * sizeof(int);

    int ha[] = {1, 2, 3, 4, 5};
    int hb[] = {10, 20, 30, 40, 50};
    int hc[cont];

    int *da, *db, *dc;
    hipMalloc(&da, size);
    hipMalloc(&db, size);
    hipMalloc(&dc, size);

    hipMemcpy(da, ha, size, hipMemcpyHostToDevice);
    hipMemcpy(db, hb, size, hipMemcpyHostToDevice);

    addArr<<<1, cont>>>(da, db, dc);
    hipMemcpy(hc, dc, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < cont; ++i)
    {
        printf("%d ", hc[i]);
    }
    return 0;
}
